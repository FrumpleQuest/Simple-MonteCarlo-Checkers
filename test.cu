#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>
#include <time.h>
#include <stdlib.h>

#define BS 256

using namespace std;


__constant__ int x_const[10000];

__global__ void kernelA(int *A, int *x, int *b, int N){
    printf("hola");
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < N*N){
        int Tid_mod_N = tid%N;
        int b_i;
        b_i = A[tid]* x[Tid_mod_N];
        atomicAdd(&b[Tid_mod_N],b_i);
    }
}

__global__ void kernelx(int *A, int *x, int *b, int N){
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < N) {
        for (int i = 0; i < N; i++){
            atomicAdd(&b[tid], A[tid + i*N] * x[tid]); //Sería mejor poner 1 atomicAdd al final, pero esto pide el enunciado
        }
    }
}

__global__ void kernelb(int *A, int *x, int *b, int N){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < N) {
		int sum = 0;
		for(int i = 0; i < N; i++) {
			sum += A[tid * N + i] * x[i];       
		}
		b[tid] = sum;
	}
}

__global__ void kernelRed(int *A, int *x, int *b, int N){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
  	__shared__ int Ps[BS];
	if (tid < N){	
		for (int row = 0; row < N; row++){
        Ps[threadIdx.x] = A[row * N + tid] * x[tid];
			__syncthreads();
			for (int thr = BS/2; thr >= 1; thr/=2){
				if (threadIdx.x < thr){
					Ps[threadIdx.x] +=  Ps[threadIdx.x + thr];
				}				
				__syncthreads();
			}
			if (threadIdx.x == 0) atomicAdd(&b[row],Ps[0]);
  		}	
	}
}


__global__ void kernelSM(int *A, int *x, int *b, int N){
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    __shared__ int x_shared_segment[BS]; 
    int suma = 0;
    if (tid < N/BS * N){ 
        for(int segment = 0; segment <= N/BS; segment++){
            x_shared_segment[threadIdx.x] = x[(threadIdx.x) + segment*BS]; 
            __syncthreads(); //Sincronizamos para evitar leer antes de escribir por completo
            if (tid < N) {
                for(int i = 0; i < BS; i++){
                    if(segment*BS + i > N-1) break; //Check de que no nos escapemos del tamaño de A
                    suma += A[i + N * tid + segment*BS] * x_shared_segment[i]; 
                }
            }
            __syncthreads();
        }
        b[tid] = suma;    
    }
}


__global__ void kernelCM(int *A, int *b, int N){
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < N) {
        int suma = 0;
        for(int i = 0; i < N; i++){
            suma += A[i + N * tid] * x_const[i];
        }
        b[tid] = suma;        
    }
}

/* ----   Codigo Principal ---- */
 

int main(int argc, char** argv) {
	
	float dt;
	int gs;
	//int bs = 256;


	/* Variables que trabajaremos */
	int N = pow(10,4);
	gs = (int)ceil((float) N*N / BS);



	int* A = new int[N*N];
	int* x = new int[N];
	int* b = new int[N];

    hipEvent_t ct1, ct2;

	//Relleno de valores
	for(int i = 0; i < N; i++){
		x[i] = 1;
		b[i] = 0; //Innecesario pero así veremos un 0 si hay errores en vez de datos basura
		for(int j = 0; j < N; j++){
			A[N*i + j] = 1;
		}
	}

    //Copiamos x a memoria constante
    hipMemcpyToSymbol(HIP_SYMBOL(x_const), x, N*sizeof(int), 0, hipMemcpyHostToDevice);

  

	/* -------- Momento GPU ----------- */

	

	
  /* Pasarle arreglos a GPU */
	
	int* AGPU;
	hipMalloc((void**)&AGPU, N*N*sizeof(int));
	hipMemcpy(AGPU, A, N*N*sizeof(int), hipMemcpyHostToDevice);

	int* xGPU;
	hipMalloc((void**)&xGPU, N*sizeof(int));
	hipMemcpy(xGPU, x, N*sizeof(int), hipMemcpyHostToDevice);

	int* bGPU;
	hipMalloc((void**)&bGPU, N*sizeof(int));
	hipMemcpy(bGPU, b, N*sizeof(int), hipMemcpyHostToDevice);
	
  
    /* ------------- Kernels -------------------- */ 

  /*  Kernel A*/
	hipEventCreate(&ct1);
	hipEventCreate(&ct2);
	hipEventRecord(ct1);

	kernelA << <gs, BS >> > (AGPU, xGPU, bGPU, N);
  
	hipEventRecord(ct2);
	hipEventSynchronize(ct2);
	hipEventElapsedTime(&dt, ct1, ct2);

	hipMemcpy(b, bGPU, N*sizeof(int), hipMemcpyDeviceToHost);
	cout << "Tiempo Kernel A: " << dt << "[ms]" << endl;
	
	
  /*  Kernel x */
	hipEventCreate(&ct1);
	hipEventCreate(&ct2);
	hipEventRecord(ct1);

	kernelx << <gs, BS >> > (AGPU, xGPU, bGPU, N);
  
	hipEventRecord(ct2);
	hipEventSynchronize(ct2);
	hipEventElapsedTime(&dt, ct1, ct2);

	hipMemcpy(b, bGPU, N*sizeof(int), hipMemcpyDeviceToHost);
	cout << "Tiempo Kernel x: " << dt << "[ms]" << endl;

	/*  Kernel b */
	hipEventCreate(&ct1);
	hipEventCreate(&ct2);
	hipEventRecord(ct1);

	kernelb << <gs, BS >> > (AGPU, xGPU, bGPU, N);
  
	hipEventRecord(ct2);
	hipEventSynchronize(ct2);
	hipEventElapsedTime(&dt, ct1, ct2);

	hipMemcpy(b, bGPU, N*sizeof(int), hipMemcpyDeviceToHost);
	cout << "Tiempo Kernel b: " << dt << "[ms]" << endl;

	/*  Kernel Red */
	hipEventCreate(&ct1);
	hipEventCreate(&ct2);
	hipEventRecord(ct1);

	kernelRed << <gs, BS >> > (AGPU, xGPU, bGPU, N);
  
	hipEventRecord(ct2);
	hipEventSynchronize(ct2);
	hipEventElapsedTime(&dt, ct1, ct2);

	hipMemcpy(b, bGPU, N*sizeof(int), hipMemcpyDeviceToHost);
	cout << "Tiempo Kernel Red: " << dt << "[ms]" << endl;


	/*  Kernel CM*/
	hipEventCreate(&ct1);
	hipEventCreate(&ct2);
	hipEventRecord(ct1);

	kernelCM << <gs, BS >> > (AGPU, bGPU, N);
  
	hipEventRecord(ct2);
	hipEventSynchronize(ct2);
	hipEventElapsedTime(&dt, ct1, ct2);

	hipMemcpy(b, bGPU, N*sizeof(int), hipMemcpyDeviceToHost);
	cout << "Tiempo Kernel CM: " << dt << "[ms]" << endl;

	/*  Kernel SM */
	hipEventCreate(&ct1);
	hipEventCreate(&ct2);
	hipEventRecord(ct1);

	kernelSM << <gs, BS >> > (AGPU, xGPU, bGPU, N);
  
	hipEventRecord(ct2);
	hipEventSynchronize(ct2);
	hipEventElapsedTime(&dt, ct1, ct2);

	hipMemcpy(b, bGPU, N*sizeof(int), hipMemcpyDeviceToHost);
	cout << "Tiempo Kernel SM: " << dt << "[ms]" << endl;

	

	
	/* Test */
	/*
    for(int i = 0; i < N; i++ ){
			if(b[i] != 10000) cout << "error" << endl;;
	}
    */

	
	/* Free */
	hipFree(AGPU); hipFree(xGPU); hipFree(bGPU); 
    delete[] A; delete[] x; delete[] b;
		
	return 0;
}