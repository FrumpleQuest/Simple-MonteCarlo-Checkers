#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>
#include <time.h>
#include <stdlib.h>
#include "funciones.h"

using namespace std;

__global__ void kernel(float* Ain, float* Aout, int M, int N, float deltax){
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < M*N){
        int tid_left = (!(tid % N)) ? (tid + N - 1) : (tid - 1);
        int tid_right = (!(tid % (N-1))) ? (tid - N + 1) : (tid + 1);
        Aout[tid] = Ain[tid] + (Ain[tid_right] - Ain[tid_left])/(2*deltax);
    }
}

/* ----   Codigo Principal ---- */
 

int main(int argc, char** argv) {

    // Variables que trabajaremos 
    int N = 8;
    int* board = new int[N*N];
    int n_fichas = 0;
    
    //Construccion de tablero inicial
    int filas_con_fichas = (N-2)/2;
    for (int i = 0; i < N*N; i++){
        board[i] = 0;
        if (i/N < filas_con_fichas){
            if ((i/N)%2){
                if (!(i%2)) board[i] = 2;
            }
            else{
                if (i%2) board[i] = 2;
            } 
        }
        else if (i/N > filas_con_fichas + 1){
            if (!(i/N%2)){
                if (i%2){
                    board[i] = 1;
                    n_fichas++;
                } 
            }
            else{
                if (!(i%2)){
                    n_fichas++;
                    board[i] = 1;
                }
            } 
        }
    }
    board[24] = 1;
    printBoard(board,N);
    int turno_jugador = 2;
    Movimientos* movimientos = generarMovimientos(board, N, n_fichas, turno_jugador);
    for(int i = 0; i < movimientos->length; i++){
        cout << movimientos->listaMovimientos[i].start_position << " " << movimientos->listaMovimientos[i].end_position << endl;
    }


    
    //Juego versión CPU


    

    bool flag_finalizado = false;
    int turno_jugador = 1; //turno_jugador 1 es del jugador
    Movimientos* movimientos;
    while(!flag_finalizado){
        printBoard(board, N);
        movimientos = generarMovimientos(board, N, n_fichas, turno_jugador);
        //Permitir a jugador escoger movimientos

        turno_jugador = (turno_jugador % 2) + 1;

        //Generar movimientos contrincante 
        //Escoger movimientos contrincante (IA)
        //if (JuegoFinalizado()) flag_finalizado = true;
    }






/*
    



    
    
	float dt;
	int gs, bs = 256;
	gs = (int)ceil((float) N*N/bs);

    
    clock_t start = clock();
    //JogodeDamas(Ain, Aout, M, N, deltax);
    clock_t end = clock();
    float seconds = (float)(end - start) / CLOCKS_PER_SEC;
    printf("Tiempo CPU: %f [ms]\n",seconds*1000);
   
    


    float* AinGPU;
    float* AoutGPU;

    hipMalloc((void**)&AinGPU, N * M * sizeof(float));
    hipMalloc((void**)&AoutGPU, N * M * sizeof(float));

    hipEvent_t e1, e2;
    hipEventCreate(&e1);
    hipEventCreate(&e2);
    hipEventRecord(e1);
    

    hipMemcpy(AinGPU, Ain, N * M * sizeof(float), hipMemcpyHostToDevice);
    kernel << <gs, bs >> > (AinGPU, AoutGPU, M, N, deltax);
    hipMemcpy(Aout, AoutGPU, M * N * sizeof(float), hipMemcpyDeviceToHost);
    Ain = Aout;
    

    hipEventRecord(e2);
    hipEventSynchronize(e2);
    hipEventElapsedTime(&dt, e1, e2);
    std::cout << "Tiempo GPU Sin Streams: " << dt << " [ms]" << std::endl;

	hipFree(AinGPU); hipFree(AoutGPU);
	delete[] Ain;
    
    */
}