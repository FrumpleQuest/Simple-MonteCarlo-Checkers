#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>
#include <time.h>
#include <stdlib.h>
#include "funciones.h"
#include <hiprand/hiprand_kernel.h>

using namespace std;


__global__ void kernel(int* board, int N, int start_position, int end_position, int kill, int n_fichas_player, int n_fichas_IA, float* evaluacion_total){
    hiprandState state;
    hiprand_init(clock64(), threadIdx.x + blockDim.x * blockIdx.x, 0, &state);

    //Creamos copia local del tablero
    int* local_board = new int[N*N];
    for(int i = 0; i < N*N; i++) local_board[i] = board[i];

    Move movimiento = {start_position,end_position, kill};
    //Aplicamos movimiento a tablero local
    execute_movement(local_board, N, movimiento, &n_fichas_player);

    //Ahora simulamos movimientos para ambos jugadores hasta que alguien gane.
    int turno_jugador = 1; //turno_jugador 1 es del jugador
    Movimientos* movimientos = new Movimientos;
    movimientos->length = 0;
    movimientos->listaMovimientos = new Move[2 * (n_fichas_IA + n_fichas_player)]; //Cantidad de movimientos es acotada
    Move player_move;
    Move IA_move;
    float winner;
    int iter = 0;
    int random;
    while(true){
            
        //Turno simulado del jugador
        movimientos = generarMovimientos(local_board, N, n_fichas_player, turno_jugador, movimientos);
        if (movimientos->length == 0){ //Si jugador se queda sin movimientos, gana la IA
            winner = 1;
            break;
        } 
        random = hiprand_uniform(&state) * movimientos->length;
        player_move = movimientos->listaMovimientos[random]; //Seleccion aleatoria de movimiento 
        execute_movement(local_board, N, player_move, &n_fichas_IA);  

        //Revisión de win condition 
        winner = win(local_board,N);
        if (winner != -1) break;
        turno_jugador = (turno_jugador % 2) + 1; 
        
            
        //Turno simulado de la IA
        movimientos = generarMovimientos(local_board, N, n_fichas_IA, turno_jugador, movimientos);
        if (movimientos->length == 0){
            winner = 0;
            break;
        } 
        random = hiprand_uniform(&state) * movimientos->length;
        IA_move = movimientos->listaMovimientos[random];
        execute_movement(local_board, N, IA_move, &n_fichas_player); 

        turno_jugador = (turno_jugador % 2) + 1;

        //Revisión de win condition 
        winner = win(local_board,N);
        if (winner != -1) break;
        iter++;
            
    }
    delete[] local_board;
    delete[] movimientos->listaMovimientos;
    delete movimientos;
    atomicAdd(evaluacion_total, winner);
}



/* ----   Codigo Principal ---- */
 

int main(int argc, char** argv) {

    // Variables que trabajaremos 
    int N = 12;
    int* board = new int[N*N];
    int n_fichas_player = 0;
    int n_fichas_IA = 0;
    char letras[] = {'A','B','C','D','E','F','G','H','K','L','M','N'};
    srand(time(NULL));
    float dt;
	int bs = 256;
	int gs = 4;
    
    //Construccion de tablero inicial
    build_board(board, N, &n_fichas_player, &n_fichas_IA);

    // Juego version CPU, IA random
    bool flag_finalizado = false;
    int turno_jugador = 1; //turno_jugador 1 es del jugador
    Movimientos* movimientos = new Movimientos;
    movimientos->length = 0;
    movimientos->listaMovimientos = new Move[2*N]; //Cantidad de movimientos es acotada
    Move player_move;
    Move IA_move;
    while(!flag_finalizado){
        //Turno del jugador
        system("clear");
        printBoard(board, N);
        movimientos = generarMovimientos(board, N, n_fichas_player, turno_jugador, movimientos);
        if (movimientos->length == 0){
            printBoard(board, N);
            printf("Ha ganado la Inteligencia Articial. La era del hombre ha llegado a su fin");
            flag_finalizado = true;
        } 
        player_move = player_select_move(movimientos, N);
        execute_movement(board, N, player_move, &n_fichas_IA);  

        //Turno de la IA
        turno_jugador = (turno_jugador % 2) + 1; 
        movimientos = generarMovimientos(board, N, n_fichas_IA, turno_jugador, movimientos);
        if (movimientos->length == 0){
            printBoard(board, N);
            printf("Ha ganado el jugador humano, venciendo a Skynet.");
            flag_finalizado = true;
        } 
        //Simulamos para cada movimiento
        int indice_maximo = 0;
        float eval_maxima = 0.;
        float* evaluacion = new float; 
        float* evaluacionGPU;
        hipMalloc((void**)&evaluacionGPU, sizeof(float));
        int* boardGPU;
        hipMalloc((void**)&boardGPU, N * N * sizeof(int));
        hipMemcpy(boardGPU, board, N * N * sizeof(int), hipMemcpyHostToDevice);
        for(int i = 0; i < movimientos->length; i++){
            *evaluacion = 0; 
            int start_position =  movimientos->listaMovimientos[i].start_position;
            int end_position =  movimientos->listaMovimientos[i].end_position;
            int kill = movimientos->listaMovimientos[i].kill;

            hipMemcpy(evaluacionGPU,evaluacion, sizeof(float), hipMemcpyHostToDevice);
            kernel << <gs, bs >> > (boardGPU, N, start_position, end_position, kill, n_fichas_player, n_fichas_IA, evaluacionGPU);
            hipMemcpy(evaluacion, evaluacionGPU, sizeof(float), hipMemcpyDeviceToHost); //Copia valor total
            
            printf("(%c%d,",letras[movimientos->listaMovimientos[i].start_position / N ], movimientos->listaMovimientos[i].start_position % N);
            printf("%c%d)",letras[movimientos->listaMovimientos[i].end_position / N ],movimientos->listaMovimientos[i].end_position % N);
            *evaluacion = (*evaluacion/(bs*gs)) * 100;
            printf("Evaluacion : %2.f%c\n",*evaluacion, '%');
            if (*evaluacion > eval_maxima){
                indice_maximo = i;
                eval_maxima = *evaluacion;
            }
        } 
        system("pause");
        IA_move = movimientos->listaMovimientos[indice_maximo];
        execute_movement(board, N, IA_move, &n_fichas_player);
        turno_jugador = (turno_jugador % 2) + 1;

        //Revisión de win condition 
        if (win(board,N) == 0){
            printf("Ha ganado el jugador humano, venciendo a Skynet.");
            flag_finalizado = true;
            
        }
        else if (win(board,N) == 1){
            printf("Ha ganado la Inteligencia Articial. La era del hombre ha llegado a su fin");
            flag_finalizado = true;
        }
    }
    
    

    /*
    hipEvent_t e1, e2;
    hipEventCreate(&e1);
    hipEventCreate(&e2);
    hipEventRecord(e1);
   

    hipEventRecord(e2);
    hipEventSynchronize(e2);
    hipEventElapsedTime(&dt, e1, e2);
    std::cout << "Tiempo GPU Sin Streams: " << dt << " [ms]" << std::endl;

	hipFree(AinGPU); hipFree(AoutGPU);
	delete[] Ain;
    */
    
}