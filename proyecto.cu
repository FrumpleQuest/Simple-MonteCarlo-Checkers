#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>
#include <time.h>
#include <stdlib.h>
#include "funciones.h"
#include <hiprand/hiprand_kernel.h>

using namespace std;


__global__ void kernel(int* board, int N, int start_position, int end_position, int kill, int n_fichas_player, int n_fichas_IA, float* evaluacion_total){
    hiprandState state;
    hiprand_init(clock64(), threadIdx.x + blockDim.x * blockIdx.x, 0, &state);

    //Creamos copia local del tablero
    int* local_board = new int[N*N];
    for(int i = 0; i < N*N; i++) local_board[i] = board[i];

    Move movimiento = {start_position,end_position, kill};
    //Aplicamos movimiento a tablero local
    execute_movement(local_board, N, movimiento, &n_fichas_player);

    //Ahora simulamos movimientos para ambos jugadores hasta que alguien gane.
    int turno_jugador = 1; //turno_jugador 1 es del jugador
    Movimientos* movimientos = new Movimientos;
    movimientos->length = 0;
    movimientos->listaMovimientos = new Move[2 * (n_fichas_IA + n_fichas_player)]; //Cantidad de movimientos es acotada
    Move player_move;
    Move IA_move;
    float winner;
    int iter = 0;
    int random;
    while(true){
            
        //Turno simulado del jugador
        movimientos = generarMovimientos(local_board, N, n_fichas_player, turno_jugador, movimientos);
        if (movimientos->length == 0){ //Si jugador se queda sin movimientos, gana la IA
            winner = 1;
            break;
        } 
        random = hiprand_uniform(&state) * movimientos->length;
        player_move = movimientos->listaMovimientos[random]; //Seleccion aleatoria de movimiento 
        execute_movement(local_board, N, player_move, &n_fichas_IA);  

        //Revisión de win condition 
        winner = win(local_board,N);
        if (winner != -1) break;
        turno_jugador = (turno_jugador % 2) + 1; 
        
            
        //Turno simulado de la IA
        movimientos = generarMovimientos(local_board, N, n_fichas_IA, turno_jugador, movimientos);
        if (movimientos->length == 0){
            winner = 0;
            break;
        } 
        random = hiprand_uniform(&state) * movimientos->length;
        IA_move = movimientos->listaMovimientos[random];
        execute_movement(local_board, N, IA_move, &n_fichas_player); 

        turno_jugador = (turno_jugador % 2) + 1;

        //Revisión de win condition 
        winner = win(local_board,N);
        if (winner != -1) break;
        iter++;
            
    }
    delete[] local_board;
    delete[] movimientos->listaMovimientos;
    delete movimientos;
    atomicAdd(evaluacion_total, winner);
}



/* ----   Codigo Principal ---- */
 
//Agregar parametros al cmd:
// N: Tamaño de tablero
// NTHREADS: Cantidad de hebras
// Verbose: 0 o 1 por si se quiere printear las probabilidades

//Arreglar warnings

//Medir Tiempos

int main(int argc, char** argv) { 

    if (argc != 4){
        printf("Porfavor ingrese 3 parametros:\n N:(>=8) NTHREADS(>0) Verbose(0|1) CPUorGPU(0|1)");
    }

    int N = atoi(argv[1]);
    int gs = atoi(argv[2]) / 256 + 1;
    int bs = atoi(argv[2]) / gs;
    int verbose = atoi(argv[3]);
    int CPUorGPU = atoi(argv[4]); //0 o 1 para CPU o GPU respectivamente

    printf("%d %d", gs,bs);

    // Variables que trabajaremos 
    int* board = new int[N*N];
    int n_fichas_player = 0;
    int n_fichas_IA = 0;
    char letras[] = {'A','B','C','D','E','F','G','H','K','L','M','N'};
    srand(time(NULL));
    float time = 0;
    clock_t t1, t2;
    
    //Construccion de tablero inicial
    build_board(board, N, &n_fichas_player, &n_fichas_IA);

    // Juego version CPU, IA random
    bool flag_finalizado = false;
    int turno_jugador = 1; //turno_jugador 1 es del jugador
    Movimientos* movimientos = new Movimientos;
    movimientos->length = 0;
    movimientos->listaMovimientos = new Move[2*N]; //Cantidad de movimientos es acotada
    Move player_move;
    Move IA_move;
    while(!flag_finalizado){
        //Turno del jugador
        system("clear");
        printf("------Turno del jugador------\n");
        printBoard(board, N);
        movimientos = generarMovimientos(board, N, n_fichas_player, turno_jugador, movimientos);
        if (movimientos->length == 0){
            printBoard(board, N);
            printf("Ha ganado la Inteligencia Articial. La era del hombre ha llegado a su fin");
            flag_finalizado = true;
        } 
        player_move = player_select_move(movimientos, N);
        execute_movement(board, N, player_move, &n_fichas_IA); 
        system("clear");
        printf("------Turno de la IA------\n");
        printBoard(board, N); 

        //Turno de la IA
        turno_jugador = (turno_jugador % 2) + 1; 
        movimientos = generarMovimientos(board, N, n_fichas_IA, turno_jugador, movimientos);
        if (movimientos->length == 0){
            printBoard(board, N);
            printf("Ha ganado el jugador humano, venciendo a Skynet.");
            flag_finalizado = true;
        } 
        //Simulamos para cada movimiento
        int indice_maximo = 0;
        float eval_maxima = 0.;
        float* evaluacion = new float; 
        float* evaluacionGPU;
        hipMalloc((void**)&evaluacionGPU, sizeof(float));
        int* boardGPU;
        hipMalloc((void**)&boardGPU, N * N * sizeof(int));
        hipMemcpy(boardGPU, board, N * N * sizeof(int), hipMemcpyHostToDevice);
        for(int i = 0; i < movimientos->length; i++){
            *evaluacion = 0; 
            int start_position =  movimientos->listaMovimientos[i].start_position;
            int end_position =  movimientos->listaMovimientos[i].end_position;
            int kill = movimientos->listaMovimientos[i].kill;

            if(CPUorGPU == 1){
                
                t1 = clock();
                hipMemcpy(evaluacionGPU,evaluacion, sizeof(float), hipMemcpyHostToDevice);
                kernel << <gs, bs >> > (boardGPU, N, start_position, end_position, kill, n_fichas_player, n_fichas_IA, evaluacionGPU);
                hipMemcpy(evaluacion, evaluacionGPU, sizeof(float), hipMemcpyDeviceToHost);
                t2 = clock();
                float ms = 1000.0 * (float)(t2 - t1) / CLOCKS_PER_SEC;
                time += ms;

            }
            else if (CPUorGPU == 0){
                t1 = clock();
                for(int j = 0; j < bs*gs; j++) *evaluacion += MonteCarloSimulation(board, N, movimientos->listaMovimientos[i], n_fichas_player, n_fichas_IA);
                t2 = clock();
                float ms = 1000.0 * (float)(t2 - t1) / CLOCKS_PER_SEC;
                time += ms;
            }

            *evaluacion = (*evaluacion/(bs*gs)) * 100;
            if (verbose == 1){
                printf("(%c%d,",letras[movimientos->listaMovimientos[i].start_position / N ], movimientos->listaMovimientos[i].start_position % N);
                printf("%c%d) ",letras[movimientos->listaMovimientos[i].end_position / N ],movimientos->listaMovimientos[i].end_position % N);
                printf("Evaluacion : %2.f%c\n",*evaluacion, '%');
                
            }           
            if (*evaluacion > eval_maxima){
                indice_maximo = i;
                eval_maxima = *evaluacion;
            }
        } 

        if (verbose == 1) system("pause");
        IA_move = movimientos->listaMovimientos[indice_maximo];
        execute_movement(board, N, IA_move, &n_fichas_player);
        turno_jugador = (turno_jugador % 2) + 1;

        //Revisión de win condition 
        if (win(board,N) == 0){
            printf("Ha ganado el jugador humano, venciendo a Skynet.\n");
            flag_finalizado = true;
            
        }
        else if (win(board,N) == 1){
            printf("Ha ganado la Inteligencia Articial. La era del hombre ha llegado a su fin.\n");
            flag_finalizado = true;
        }
    }
    if (CPUorGPU == 0) printf("Tiempo de computo medido por CPU fue: %f [ms]\n", time);
    else printf("Tiempo de computo medido por GPU fue: %f [ms]\n", time);
    

    


	//hipFree(AinGPU); hipFree(AoutGPU);
	//delete[] Ain;
    
    
}