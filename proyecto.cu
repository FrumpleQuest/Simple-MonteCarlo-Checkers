

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
using namespace std;

struct Nodo{
    int* solucion;
    Nodo* izq;
    Nodo* der;
};

void printBoard(int* board, int N){
    for (int j = 0; j < N; j++){
        for(int i = 0; i < N; i++){
            cout << board[i +  j*N] << " ";
        }
        cout << endl;
    }
}

//Funcion de evaluación (o quizas se gana instantaneamente al llegar a la fila enemiga)
float eval(int* board, int N);

//Función que retorna una lista con movimientos

//Funcion que escoge un movimiento al azar


int main(){
    //Versión simple de juego de damas
    int N = 16;
    int* board = new int[N*N];
    
    int filas_con_fichas = (N-2)/2;
    //Construccion de tablero inicial
    for (int i = 0; i < N*N; i++){
        board[i] = 0;
        if (i/N < filas_con_fichas){ //Hardcodeado el 3, podría depender de N para N damas.
            if ((i/N)%2){
                if (!(i%2)) board[i] = 1;
            }
            else{
                if (i%2) board[i] = 1;
            } 
        }
        else if (i/N > filas_con_fichas + 1){
            if (!(i/N%2)){
                if (i%2) board[i] = 2;
            }
            else{
                if (!(i%2)) board[i] = 2;
            } 
        }
    }
    printBoard(board, N);






}