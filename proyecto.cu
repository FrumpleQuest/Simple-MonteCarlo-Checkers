#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>
#include <time.h>
#include <stdlib.h>
#include "funciones.h"
#include <hiprand/hiprand_kernel.h>

__global__ void kernel(int* board, int N, int NTHREADS , Move movimiento, int move_number, /*int start_position, int end_position, int kill,*/ int n_fichas_player, int n_fichas_IA, float* evaluacionGPU, float* evalesGPU){
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    
    if (tid < NTHREADS){
        hiprandState state;
        hiprand_init(clock64(), threadIdx.x + blockDim.x * blockIdx.x, 0, &state);

        //Creamos copia local del tablero
        int* local_board = new int[N*N];
        for(int i = 0; i < N*N; i++) local_board[i] = board[i];

        //Move movimiento = {start_position,end_position, kill};
        //Aplicamos movimiento a tablero local
        execute_movement(local_board, N, movimiento, &n_fichas_player);

        //Ahora simulamos movimientos para ambos jugadores hasta que alguien gane.
        int turno_jugador = 1; //turno_jugador 1 es del jugador
        Movimientos* movimientos = new Movimientos;
        movimientos->length = 0;
        movimientos->listaMovimientos = new Move[2 * (n_fichas_IA + n_fichas_player)]; //Cantidad de movimientos es acotada
        Move player_move;
        Move IA_move;
        float winner;
        int iter = 0;
        int random;
        while(true){
                
            //Turno simulado del jugador
            movimientos = generarMovimientos(local_board, N, n_fichas_player, turno_jugador, movimientos);
            if (movimientos->length == 0){ //Si jugador se queda sin movimientos, gana la IA
                winner = 1;
                break;
            } 
            random = hiprand_uniform(&state) * movimientos->length;
            player_move = movimientos->listaMovimientos[random]; //Seleccion aleatoria de movimiento 
            execute_movement(local_board, N, player_move, &n_fichas_IA);  

            //Revisión de win condition 
            winner = win(local_board,N);
            if (winner != -1) break;
            turno_jugador = (turno_jugador % 2) + 1; 
            
                
            //Turno simulado de la IA
            movimientos = generarMovimientos(local_board, N, n_fichas_IA, turno_jugador, movimientos);
            if (movimientos->length == 0){
                winner = 0;
                break;
            } 
            random = hiprand_uniform(&state) * movimientos->length;
            IA_move = movimientos->listaMovimientos[random];
            execute_movement(local_board, N, IA_move, &n_fichas_player); 

            turno_jugador = (turno_jugador % 2) + 1;

            //Revisión de win condition 
            winner = win(local_board,N);
            if (winner != -1) break;
            iter++;
                
        }
        delete[] local_board;
        delete[] movimientos->listaMovimientos;
        delete movimientos;
        atomicAdd(&evalesGPU[move_number], winner);
        printf("evaluacion: %f\n", evalesGPU[move_number]);
   }
    
}



/* ----   Codigo Principal ---- */
 
//Agregar parametros al cmd:
// N: Tamaño de tablero
// NTHREADS: Cantidad de hebras
// Verbose: 0 o 1 por si se quiere printear las probabilidades


//Medir Tiempos

int main(int argc, char** argv) { 

    if (argc != 5){
        printf("Porfavor ingrese 4 parametros:\n N:(>=8) NTHREADS(>0) Verbose(0|1) CPUorGPU(0|1)");
    }

    //int N = atoi(argv[1]);
    int N =8;
    //int NTHREADS = atoi(argv[2]);
    int NTHREADS = 100;
    //int verbose = atoi(argv[3]);
    int verbose = 1;
    //int CPUorGPU = atoi(argv[4]); //0 o 1 para CPU o GPU respectivamente
    int CPUorGPU = 1;
    
    int bs = 256;
    int gs = (int)ceil((float) NTHREADS / bs);

    printf("%d %d\n", gs,bs);

    // Variables que trabajaremos 
    int* board = new int[N*N];
    int n_fichas_player = 0;
    int n_fichas_IA = 0;
    char letras[] = {'A','B','C','D','E','F','G','H','K','L','M','N'};
    srand(time(NULL));
    float time = 0;
    clock_t t1, t2;
    
    //Construccion de tablero inicial
    build_board(board, N, &n_fichas_player, &n_fichas_IA);

    // Juego version CPU, IA random
    bool flag_finalizado = false;
    int turno_jugador = 1; //turno_jugador 1 es del jugador
    Movimientos* movimientos = new Movimientos;
    movimientos->length = 0;
    movimientos->listaMovimientos = new Move[2*N]; //Cantidad de movimientos es acotada
    Move player_move;
    Move IA_move;
    while(!flag_finalizado){
        //Turno del jugador
        system("clear");
        printf("------Turno del jugador------\n");
        printBoard(board, N);
        movimientos = generarMovimientos(board, N, n_fichas_player, turno_jugador, movimientos);
        if (movimientos->length == 0){
            printBoard(board, N);
            printf("Ha ganado la Inteligencia Articial. La era del hombre ha llegado a su fin");
            flag_finalizado = true;
        } 
        player_move = player_select_move(movimientos, N);
        execute_movement(board, N, player_move, &n_fichas_IA); 
        system("clear");
        

        //Turno de la IA
        printf("------Turno de la IA------\n");
        printBoard(board, N);
        turno_jugador = (turno_jugador % 2) + 1; 
        movimientos = generarMovimientos(board, N, n_fichas_IA, turno_jugador, movimientos);
        if (movimientos->length == 0){
            printBoard(board, N);
            printf("Ha ganado el jugador humano, venciendo a Skynet.");
            flag_finalizado = true;
        } 
       
        //Simulamos para cada movimiento
        int indice_maximo = 0;
        float eval_maxima = 0.;
        float* evaluacion = new float; 
        
       /* GPU MODE */
        if (CPUorGPU == 1){
            
            float* evalesCPU = new float[movimientos->length];
            for(int i = 0; i < movimientos->length; i++ ) evalesCPU[i] = 0;  

            float* evalesGPU;
            hipMalloc((void**)&evalesGPU, movimientos->length * sizeof(float));
            
            float* evaluacionGPU;
            hipMalloc((void**)&evaluacionGPU, sizeof(float));

            int* boardGPU;
            hipMalloc((void**)&boardGPU, N * N * sizeof(int));
            hipMemcpy(boardGPU, board, N * N * sizeof(int), hipMemcpyHostToDevice);

            hipEvent_t ct1, ct2;
            const int nstreams = 2;
            hipStream_t streams[nstreams];
            int streamsize = movimientos->length/nstreams;

            
            
            for(int i = 0; i < streamsize  ; i++){             
                for (int j = 0; j < nstreams; j++ ){
                    
                    *evaluacion = 0;
                    int move_number = j + i*nstreams;
                    hipStreamCreate(&streams[j]);

                    hipMemcpyAsync(evaluacionGPU, evaluacion,  sizeof(float), hipMemcpyHostToDevice, streams[j]);
                    hipMemcpyAsync(evalesGPU, evalesCPU,    movimientos->length * sizeof(float), hipMemcpyHostToDevice, streams[j]);
                    kernel <<< gs, bs, 0, streams[j] >>> (boardGPU, N, NTHREADS, movimientos->listaMovimientos[move_number], move_number, /*start_position, end_position, kill,*/ n_fichas_player, n_fichas_IA, evaluacionGPU, evalesGPU);
                    hipMemcpyAsync(evalesCPU, evalesGPU,  movimientos->length  * sizeof(float), hipMemcpyDeviceToHost, streams[j]);
                   
                } 

                hipDeviceSynchronize();

                //float ms = 1000.0 * (float)(t2 - t1) / CLOCKS_PER_SEC;
                //time += ms;
                
            }

            if(movimientos->length%nstreams != 0){
                *evaluacion = 0;
                int move_number = (movimientos->length)-1;
                hipMemcpy(evaluacionGPU, evaluacion, sizeof(float), hipMemcpyHostToDevice);
                hipMemcpy(evalesGPU, evalesCPU, movimientos->length * sizeof(float), hipMemcpyHostToDevice);
                kernel <<< gs, bs>>> (boardGPU, N, NTHREADS, movimientos->listaMovimientos[move_number],  move_number, /* start_position, end_position, kill,*/ n_fichas_player, n_fichas_IA, evaluacionGPU, evalesGPU);
                hipMemcpy(evalesCPU, evalesGPU, movimientos->length * sizeof(int), hipMemcpyDeviceToHost);
            }

            

            for (int i = 0; i <  movimientos->length ; i++){
               if (evalesCPU[i] > eval_maxima){
                    indice_maximo = i;
                    eval_maxima = evalesCPU[i];
                }
                if (verbose == 1){
                    printf("(%c%d,",letras[movimientos->listaMovimientos[i].start_position / N ], movimientos->listaMovimientos[i].start_position % N);
                    printf("%c%d) ",letras[movimientos->listaMovimientos[i].end_position / N ],movimientos->listaMovimientos[i].end_position % N);
                    printf("Evaluacion : %2.f %c\n", (evalesCPU[i]/ NTHREADS) * 100 , '%');
                    
                 } 
            }

            hipFree(evaluacionGPU); hipFree(boardGPU); hipFree(evalesGPU); 
            delete[] evaluacion; delete[] evalesCPU;
            
        }

        
        
        /* CPU MODE */
        else if (CPUorGPU == 0) {
            for(int i = 0; i < movimientos->length; i++){
                *evaluacion = 0; 
                t1 = clock();
                for(int j = 0; j < NTHREADS; j++) *evaluacion += MonteCarloSimulation(board, N, movimientos->listaMovimientos[i], n_fichas_player, n_fichas_IA);
                t2 = clock();
                float ms = 1000.0 * (float)(t2 - t1) / CLOCKS_PER_SEC;
                time += ms;

                *evaluacion = (*evaluacion/(NTHREADS)) * 100;
                
                if (*evaluacion > eval_maxima){
                    indice_maximo = i;
                    eval_maxima = *evaluacion;
                }

                 if (verbose == 1){
                    printf("(%c%d,",letras[movimientos->listaMovimientos[i].start_position / N ], movimientos->listaMovimientos[i].start_position % N);
                    printf("%c%d) ",letras[movimientos->listaMovimientos[i].end_position / N ],movimientos->listaMovimientos[i].end_position % N);
                    printf("Evaluacion : %2.f%c\n",*evaluacion, '%');
                 }
                
             }
             delete[] evaluacion;
        }

        if (verbose == 1) system("pause");

         
        /* IA ejecuta movimiento */
        IA_move = movimientos->listaMovimientos[indice_maximo];
        execute_movement(board, N, IA_move, &n_fichas_player);
        turno_jugador = (turno_jugador % 2) + 1;

        //Revisión de win condition 
        if (win(board,N) == 0){
            printf("Ha ganado el jugador humano, venciendo a Skynet.\n");
            flag_finalizado = true;
            
        }
        else if (win(board,N) == 1){
            printf("Ha ganado la Inteligencia Articial. La era del hombre ha llegado a su fin.\n");
            flag_finalizado = true;
        }
    }
    
    if (CPUorGPU == 0) printf("Tiempo de computo medido por CPU fue: %f [ms]\n", time);
    else printf("Tiempo de computo medido por GPU fue: %f [ms]\n", time);
      
}